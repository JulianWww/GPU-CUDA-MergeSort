
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#include <stdint.h>
#include <chrono>
#include <vector>

bool IsNotPowerOfTwo(const ulong x)
{
    return (x & (x - 1)) != 0;
}

static inline uint32_t getSteps(const uint32_t x) {
  uint32_t y;
  asm ( "\tbsr %1, %0\n"
      : "=r"(y)
      : "r" (x)
  );
  return y + (int)IsNotPowerOfTwo(x);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__
void GPUsortStep(int* x, int* y, int n, int sortSize)
{
    int i = (blockIdx.x*blockDim.x + threadIdx.x)*sortSize*2;
    int maxIter = i + sortSize*2;
    int maxAIter = i + sortSize;
    if (i < n){
        if (n < maxIter)
        {
            maxIter = n;
        }
        if (n <= maxAIter)
        {
            while (i < n)
            {
                y[i] = x[i];
                i ++;
            }
            return;
        }

        int iter = i;
        int iterA = i;
        int iterB = i + sortSize;
        if (iterB < n) 
        {
            while (iter < maxIter)
            {
                if (iterA >= maxAIter)
                {
                    y[iter] = x[iterB];
                    iterB ++;
                }
                else if (iterB >= maxIter)
                {
                    y[iter] = x[iterA];
                    iterA++;
                }
                else
                {
                    if (x[iterA] > x[iterB])
                    {
                        y[iter] = x[iterB];
                        iterB ++;
                    }
                    else
                    {
                        y[iter] = x[iterA];
                        iterA ++;
                    }
                }
                iter++;
            }
        }
    }
}

void CPUsortStep(int* x, int* y, int n, int sortSize, int i)
{
    int maxIter = i + sortSize*2;
    int maxAIter = i + sortSize;
    if (n < maxIter)
    {
        maxIter = n;
    }
    if (n <= maxAIter)
    {
        while (i < n)
        {
            y[i] = x[i];
            i ++;
        }
        return;
    }
    int iter = i;
    int iterA = i;
    int iterB = i + sortSize;
    if (iterB < n) 
    {
        while (iter < maxIter)
        {
            if (iterA >= maxAIter)
            {
                y[iter] = x[iterB];
                iterB ++;
            }
            else if (iterB >= maxIter)
            {
                y[iter] = x[iterA];
                iterA++;
            }
            else
            {
                if (x[iterA] > x[iterB])
                {
                    y[iter] = x[iterB];
                    iterB ++;
                }
                else
                {
                    y[iter] = x[iterA];
                    iterA ++;
                }
            }
            iter++;
        }
    }
}

void cudaSort(int * x,  int n, int steps, int& sorterSize)
{
    int *cuda_x, *cuda_y;
    gpuErrchk(hipMalloc(&cuda_x, n*sizeof(int)));
    gpuErrchk(hipMalloc(&cuda_y, n*sizeof(int)));

    gpuErrchk(hipMemcpy(cuda_x, x, n*sizeof(int), hipMemcpyHostToDevice));

    sorterSize = 1;
    for (size_t idx = 0; idx < steps; idx++)
    {
        const unsigned int threads = n / sorterSize;
        std::cout << idx << "\t" << threads << std::endl;
        if (threads < 256)
        {
            GPUsortStep<<<1, threads>>>(cuda_x, cuda_y, n, sorterSize);
        }
        else
        {
            GPUsortStep<<<(threads + 255)/256, 256>>>(cuda_x, cuda_y, n, sorterSize);
        }
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        sorterSize = sorterSize * 2;
        std::swap(cuda_x, cuda_y);
    }

    gpuErrchk(hipMemcpy(x, cuda_x, n*sizeof(int), hipMemcpyDeviceToHost));

    gpuErrchk(hipFree(cuda_x));
    gpuErrchk(hipFree(cuda_y));
}

int * cpuSort(int * x, int n, int steps, int size=10)
{
    int* y = (int*)malloc(sizeof(int)*n);
    for (size_t idx = 0; idx < steps; idx++)
    {
        size_t i = 0;
        while (i < n)
        {
            CPUsortStep(x, y, n, size, i);
            i += 2*size;
        }
        std::swap(x,y);
        size = size*2;
    }
    free(y);
    return x;
}

int * sort(int* x, int n, int cpuSteps = 10)
{
    int *out = (int*)malloc(n*sizeof(int));
    int sorterSize;
    auto steps = getSteps(n);
    cudaSort(x, n, steps-cpuSteps, sorterSize);
    return cpuSort(x, n, cpuSteps, sorterSize);
}

size_t test(int nums, int size)
{
  
  int N = 1<<size;
  std::cout << N * sizeof(int) << std::endl;
  int *x, *d_x, *d_y;
  x = (int*)malloc(N*sizeof(int));

  //cudaMalloc(&d_x, N*sizeof(int));
  //cudaMalloc(&d_y, N*sizeof(int)); 

  for (int i = 0; i < N; i++) {
    x[i] = std::rand();
  }
  std::cout << std::endl;

  auto start = std::chrono::high_resolution_clock::now();
  x = sort(x, N, nums);
  auto stop = std::chrono::high_resolution_clock::now();
  size_t timeTaken = std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count();

  for (int i = 0; i+1 < N; i++)
  {
    if (x[i+1] < x[i])
    {
        printf("error\n");
        return 1;
    }
    if(x[i] == 0){
        printf("NULL\n");
        return 1;
    }
  }

  free(x);

  std::cout << "sorting took: " << timeTaken << " ns" << std::endl;
  return timeTaken;
}

int main(void)
{
    std::vector<size_t> times = {};
    int N = 27;
    for (int idx = 0; idx < N; idx++)
    {
        times.push_back(test(idx, N));
    }
    for (auto const& val : times)
    {
        std::cout << val << ", ";
    }
    std::cout << std::endl;
}